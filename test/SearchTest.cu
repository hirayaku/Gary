#include "hip/hip_runtime.h"
#include <random>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "HelperCUDA.h"
#include "gtest/gtest.h"
#include "Timer.h"
#include "ops/DeviceIterator.cuh"
#include "ops/Search.cuh"

template <typename T>
__global__ void binarySearchKernel(T* data, size_t len, T* keys, size_t numKeys,
  bool* results) {

  const auto thisGrid = cg::this_grid();
  const auto thisBlock = cg::this_thread_block();
  const auto threadId = thisGrid.thread_rank();

  Span<T*, void> list {data, len};
  for (auto idx : IdRange<size_t, decltype(thisGrid)> {0, numKeys, thisGrid}) {
    results[idx] = binary_search(list, keys[idx]);
  }
}

template <typename T>
__global__ void binarySearchCountKernel(T* data, size_t len, T* keys, size_t numKeys,
  int* count) {

  const auto thisGrid = cg::this_grid();
  const auto thisBlock = cg::this_thread_block();
  const auto threadId = thisGrid.thread_rank();
  // __shared__ int blockCount[1];

  Span<T*, void> list {data, len};
  for (auto idx : IdRange<size_t, decltype(thisGrid)> {0, numKeys, thisGrid}) {
    if (binary_search(list, keys[idx]))
      atomicAdd(count, 1);
  }
  // // block-level atomic add
  // thisBlock.sync();
  // if (thisBlock.thread_rank() == 0) {
  //   atomicAdd(count, blockCount[0]);
  // }
}

template <typename T, size_t N=32>
__global__ void binarySearch2PhaseKernel(T* data, size_t len, T* keys, size_t numKeys,
  bool* results) {

  extern __shared__ T smem[];
  const auto thisGrid = cg::this_grid();
  const auto thisBlock = cg::this_thread_block();
  const auto thisWarp = cg::tiled_partition<WARP_SIZE>(thisBlock);
  const auto warpIdBlock = thisWarp.meta_group_rank();

  Span<T*, void> list {data, len};
  Array<T, N> warpCache {smem + warpIdBlock * N};
  build_cache(list, warpCache, thisWarp);
  thisWarp.sync();

  for (auto idx : IdRange<size_t, decltype(thisGrid)> {0, numKeys, thisGrid}) {
    results[idx] = binary_search_2phase(list, warpCache, keys[idx]);
  }
}

template <typename T, size_t N=32>
__global__ void binarySearch2PhaseCountKernel(T* data, size_t len, T* keys, size_t numKeys,
  int* count) {

  // __shared__ int blockCount[WARP_SIZE];
  extern __shared__ T smem[];
  const auto thisGrid = cg::this_grid();
  const auto thisBlock = cg::this_thread_block();
  const auto thisWarp = cg::tiled_partition<WARP_SIZE>(thisBlock);
  const auto warpIdBlock = thisWarp.meta_group_rank();
  const auto threadIdBlock = thisBlock.thread_rank();

  Span<T*, void> list {data, len};
  Array<T, N> warpCache {smem + warpIdBlock * N};
  build_cache(list, warpCache, thisWarp);
  thisWarp.sync();

  int *blockCount = smem + thisWarp.meta_group_size() * N;

  for (auto idx : IdRange<size_t, decltype(thisGrid)> {0, numKeys, thisGrid}) {
    if (binary_search_2phase(list, warpCache, keys[idx])) {
      atomicAdd_block(count, 1);
    }
  }
  // thisBlock.sync();
  // if (threadIdBlock == 0)
  //   atomicAdd(count, *blockCount);
}

template <typename T> __global__ void
binarySearch2PhaseOG(T *search, size_t search_size, T *lookup, size_t lookup_size,
  bool* results) {

  //if (size_a == 0 || size_b == 0) return 0;
  int thread_lane = threadIdx.x & (WARP_SIZE-1); // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;     // warp index within the CTA
  // __shared__ T cache[BLOCK_SIZE];
  extern __shared__ T cache[];
  cache[warp_lane * WARP_SIZE + thread_lane] = search[thread_lane * search_size / WARP_SIZE];
  __syncwarp();

  for (auto i = thread_lane; i < lookup_size; i += WARP_SIZE) {
    T key = lookup[i]; // each thread picks a vertex as the key
    results[i]  = binary_search_2phase(search, cache, key, (int) search_size);
  }
}

template <typename T, typename CudaGroup>
__global__ void linearSearchKernel(T* data, size_t len, T* keys, size_t numKeys,
  bool* results) {
  // TODO
}

class SearchTest : public testing::Test {
protected:
  static std::mt19937 gen;
  static thrust::host_vector<int> testSearch;
  static thrust::host_vector<int> testKeys;
  // TODO: there's some bug when sizes are above this value
  static constexpr int cacheSize = 32; 

  static void SetUpTestCase() {
    std::random_device rd;
    int seed = rd();
    gen = std::mt19937(seed);
    GTEST_LOG_(INFO) << "Using seed=" << seed;

    int deviceCount = 0;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));
    gpuDeviceInit(0);

    // generate common test data
    constexpr int low = 1024 * 1024 * 1024 / 2, high = 1024 * 1024 * 1024;
    const int keyLow = low, keyHigh = high;
    constexpr int numItems = 1024 * 1024 * 16;
    constexpr int numKeys = 1024 * 1024 + 1;
    testSearch = GenerateRandomIntegers(numItems, low, high);
    testKeys = GenerateRandomIntegers(numKeys, keyLow, keyHigh);
  }

  // static void TearDownTestCase();

  template <typename T> static thrust::host_vector<T>
  GenerateRandomIntegers(size_t len, T min, T max) {
    std::uniform_int_distribution<T> dis(min, max);
    thrust::host_vector<T> result(len);
    for (size_t i = 0; i < len; ++i)
      result[i] = dis(gen);
    return result;
  }

  template <typename T> static thrust::host_vector<bool>
  RunStdBinarySearch(
    const thrust::host_vector<T> &vec, const thrust::host_vector<T> &search
  ) {
    thrust::host_vector<bool> results(search.size());
    for (size_t i = 0; i < search.size(); ++i) {
      results[i] = std::binary_search(vec.begin(), vec.end(), search[i]);
    }
    return results;
  }

  template <typename T> static thrust::host_vector<bool>
  RunCPUBinarySearch(
    const thrust::host_vector<T> &vec, const thrust::host_vector<T> &search
  ) {
    thrust::host_vector<bool> results(search.size());
    Span<const T*, void> list {thrust::raw_pointer_cast(&vec[0]), vec.size()};
    for (size_t i = 0; i < search.size(); ++i)
      results[i] = binary_search(list, search[i]);
    return results;
  }

  template <typename T, size_t N> static thrust::host_vector<bool>
  RunCPUBinarySearch2Phase(
    const thrust::host_vector<T> &vec, const thrust::host_vector<T> &keys,
    const Array<T, N> &cache
  ) {
    thrust::host_vector<bool> results(keys.size());
    Span<const T*, void> list {thrust::raw_pointer_cast(vec.data()), vec.size()};
    for (size_t i = 0; i < keys.size(); ++i) {
      results[i] = binary_search_2phase<const T>(list, cache, keys[i]);
    }
    return results;
  }

  template <typename T> static thrust::host_vector<bool>
  RunCUDABinarySearch(
    const thrust::host_vector<T> &h_in, const thrust::host_vector<T> &h_search,
    dim3 gridDim, dim3 blockDim, utils::CUDATimer &timer, bool count) {

    thrust::device_vector<T> d_in = h_in;
    thrust::device_vector<T> d_search = h_search;
    thrust::device_vector<bool> d_results(h_search.size());

    timer.start();
    if (count) {
      binarySearchCountKernel<T> <<<gridDim, blockDim, 0, timer.stream()>>> (
        thrust::raw_pointer_cast(d_in.data()), d_in.size(),
        thrust::raw_pointer_cast(d_search.data()), d_search.size(),
        (int *)thrust::raw_pointer_cast(d_results.data()));
    } else {
      binarySearchKernel<T> <<<gridDim, blockDim, 0, timer.stream()>>> (
        thrust::raw_pointer_cast(d_in.data()), d_in.size(),
        thrust::raw_pointer_cast(d_search.data()), d_search.size(),
        thrust::raw_pointer_cast(d_results.data()));
    }
    timer.stop();
    checkCudaErrors(hipGetLastError());

    thrust::host_vector<bool> h_results = d_results;
    return h_results;
  }

  template <typename T, size_t N> static thrust::host_vector<bool>
  RunCUDABinarySearch2Phase(
    const thrust::host_vector<T> &h_in, const thrust::host_vector<T> &h_search,
    dim3 gridDim, dim3 blockDim, utils::CUDATimer &timer, bool count) {

    thrust::device_vector<T> d_in = h_in;
    thrust::device_vector<T> d_search = h_search;
    thrust::device_vector<bool> d_results(h_search.size(), 0);
    size_t smem = N * sizeof(T) * warpsPerBlock(blockDim);

    timer.start();
    if (count) {
      binarySearch2PhaseCountKernel<T, N> <<<gridDim, blockDim, smem+4, timer.stream()>>> (
        thrust::raw_pointer_cast(d_in.data()), d_in.size(),
        thrust::raw_pointer_cast(d_search.data()), d_search.size(),
        (int *)thrust::raw_pointer_cast(d_results.data()));
    } else {
      binarySearch2PhaseKernel<T, N> <<<gridDim, blockDim, smem, timer.stream()>>> (
        thrust::raw_pointer_cast(d_in.data()), d_in.size(),
        thrust::raw_pointer_cast(d_search.data()), d_search.size(),
        thrust::raw_pointer_cast(d_results.data()));
    }
    timer.stop();
    checkCudaErrors(hipGetLastError());

    thrust::host_vector<bool> h_results = d_results;
    return h_results;
  }
};

std::mt19937 SearchTest::gen;
thrust::host_vector<int> SearchTest::testSearch {};
thrust::host_vector<int> SearchTest::testKeys {};

TEST_F(SearchTest, BinarySearchCPU) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  utils::Timer timer;
  timer.start();
  const auto results = RunCPUBinarySearch(vec, keys);
  timer.stop();

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    EXPECT_TRUE(results[i] == resultsRef[i]) << " at index: " << i << "\n";
}

TEST_F(SearchTest, BinarySearchWithCacheCPU) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  utils::Timer timer;
  timer.start();

  thrust::host_vector<int> cache(cacheSize);
  Span<const int*, void> list {thrust::raw_pointer_cast(vec.data()), vec.size()};
  Array<int, cacheSize> cacheView {cache.data()};
  build_cache(list, cacheView);

  // printf("list:  min: %d, max: %d\n", vec[0], vec[vec.size()-1]);
  // printf("keys:  min: %d, max: %d\n", keys[0], keys[keys.size()-1]);
  // printf("cache: [");
  // for (int i = 0; i < cacheSize; ++i)
  //   printf("%d ", cache[i]);
  // printf("]\n");

  const auto results = RunCPUBinarySearch2Phase(vec, keys, cacheView);
  timer.stop();
  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    ASSERT_EQ(results[i], resultsRef[i]) << " at index: " << i
      << " item: " << keys[i] << "\n";
}

TEST_F(SearchTest, BinarySearchCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA", stream);
  dim3 gridDim(8);
  dim3 blockDim(512);
  const auto results = RunCUDABinarySearch<int>(vec, keys, gridDim, blockDim, timer, false);

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    ASSERT_EQ(results[i], resultsRef[i]) << " at index: " << i
      << " item: " << keys[i] << "\n";
}

// 2-phase binary search doesn't give speedups compared to the straightforward one,
// when the key range falls within the search range
TEST_F(SearchTest, BinarySearchWithCacheCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA (cache)", stream);
  dim3 gridDim(8);
  dim3 blockDim(512);
  const auto results = RunCUDABinarySearch2Phase<int, cacheSize>(
    vec, keys, gridDim, blockDim, timer, false);

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    ASSERT_EQ(results[i], resultsRef[i]) << " at index: " << i
      << " item: " << keys[i] << "\n";
}

TEST_F(SearchTest, BinarySearchOriginalCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  thrust::device_vector<int> d_in = vec;
  thrust::device_vector<int> d_keys = keys;
  thrust::device_vector<bool> d_results(keys.size(), 0);

  dim3 gridDim(1);
  dim3 blockDim(32);
  // original method use an index cache of WARP_SIZE
  size_t smem = WARP_SIZE * sizeof(int) * warpsPerBlock(blockDim);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA (cache)", stream);
  timer.start();
  binarySearch2PhaseOG<int> <<<gridDim, blockDim, smem, timer.stream()>>> (
    thrust::raw_pointer_cast(d_in.data()), d_in.size(),
    thrust::raw_pointer_cast(d_keys.data()), d_keys.size(),
    thrust::raw_pointer_cast(d_results.data()));
  timer.stop();
  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  thrust::host_vector<bool> results = d_results;
  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    ASSERT_EQ(results[i], resultsRef[i]) << " at index: " << i
      << " item: " << keys[i] << "\n";
}

TEST_F(SearchTest, BinarySearchCountCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);
  const auto countRef = std::accumulate(resultsRef.begin(), resultsRef.end(), 0);
  GTEST_LOG_(INFO) << countRef << " items found";

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA", stream);
  dim3 gridDim(8);
  dim3 blockDim(512);
  const auto results = RunCUDABinarySearch<int>(vec, keys, gridDim, blockDim, timer, true);

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(*(int *)results.data(), countRef);
}

// 2-phase binary search doesn't give speedups compared to the straightforward one,
// when the key range falls within the search range
TEST_F(SearchTest, BinarySearchWithCacheCountCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  std::sort(keys.begin(), keys.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);
  const auto countRef = std::accumulate(resultsRef.begin(), resultsRef.end(), 0);
  GTEST_LOG_(INFO) << countRef << " items found";

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA (cache)", stream);
  dim3 gridDim(8);
  dim3 blockDim(512);
  const auto results = RunCUDABinarySearch2Phase<int, cacheSize>(
    vec, keys, gridDim, blockDim, timer, true);

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(*(int *)results.data(), countRef);
}

TEST_F(SearchTest, BinarySearchUnsortedCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA", stream);
  dim3 gridDim(8);
  dim3 blockDim(512);
  const auto results = RunCUDABinarySearch<int>(vec, keys, gridDim, blockDim, timer, false);

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    ASSERT_EQ(results[i], resultsRef[i]) << " at index: " << i
      << " item: " << keys[i] << "\n";
}

// 2-phase binary search doesn't give speedups compared to the straightforward one,
// when keys are unsorted
TEST_F(SearchTest, BinarySearchWithCacheUnsortedCUDA) {
  auto vec = testSearch;
  auto keys = testKeys;
  std::sort(vec.begin(), vec.end());
  const auto resultsRef = RunStdBinarySearch(vec, keys);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));
  utils::CUDATimer timer("Binary search on CUDA (cache)", stream);
  dim3 gridDim(8);
  dim3 blockDim(512);
  const auto results = RunCUDABinarySearch2Phase<int, cacheSize>(
    vec, keys, gridDim, blockDim, timer, false);

  GTEST_LOG_(INFO) << "target takes " << timer.microsecs() << "us";

  ASSERT_EQ(results.size(), resultsRef.size());
  for (size_t i = 0; i < results.size(); ++i)
    ASSERT_EQ(results[i], resultsRef[i]) << " at index: " << i
      << " item: " << keys[i] << "\n";
}
